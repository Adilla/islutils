#include <assert.h>
#include <stdio.h>
#include "gemm_kernel.hu"
int D[1024][1024];
int C[1024][1024];
int A[1024][1024];
int B[1024][1024];
int alpha, beta;

void gemm() {

  for (int i = 0; i < 1024; i++)
    for (int j = 0; j < 1024; j++) 
        D[i][j] = beta * C[i][j];

  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

    hipblasHandle_t handle;
    int *dev_A;
    int *dev_B;
    int *dev_D;
    
    cudaCheckReturn(hipblasCreate(&handle));
    cudaCheckReturn(hipMalloc((void **) &dev_A, (1024) * (1024) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_B, (1024) * (1024) * sizeof(int)));
    cudaCheckReturn(hipMalloc((void **) &dev_D, (1024) * (1024) * sizeof(int)));
    
    cudaCheckReturn(hipblasSetMatrix(1024, 1024, sizeof(int), A, 1024, dev_A, 1024));
    cudaCheckReturn(hipblasSetMatrix(1024, 1024, sizeof(int), B, 1024, dev_B, 1024));
    cudaCheckReturn(hipblasSetMatrix(1024, 1024, sizeof(int), D, 1024, dev_D, 1024));
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(32, 32);
      kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dev_A, dev_B, dev_D, alpha);
      cudaCheckKernel();
    }
    
    cudaCheckReturn(cudaGetMatrix(1024, 1024, sizeof(int), dev_D, 1024, D, 1024));
    cudaCheckReturn(hipFree(dev_A));
    cudaCheckReturn(hipFree(dev_B));
    cudaCheckReturn(hipFree(dev_D));
    cudaCheckReturn(hipblasDestroy(handle));
  }
}
