#include "hip/hip_runtime.h"
#include "tmm_including_transpose_loop_kernel.hu"
__global__ void kernel0(int *B, int alpha)
{
    int b0 = blockIdx.x;
    int t0 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c1 = ppcg_max(0, 32 * b0 - 1024); c1 <= ppcg_min(1023, 32 * b0 + 31); c1 += 32) {
      for (int c3 = ppcg_max(0, 32 * b0 + t0 - c1 - 1023); c3 <= ppcg_min(31, 32 * b0 + t0 - c1); c3 += 1)
        B[(c1 + c3) * 1024 + (32 * b0 + t0 - c1 - c3)] = B[(32 * b0 + t0 - c1 - c3) * 1024 + (c1 + c3)];
      __syncthreads();
    }
}
__global__ void kernel1(int *A, int *B, int *D, int alpha)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ int shared_A[32][32];
    __shared__ int shared_B[32][32];
    int private_D[1][2];

    {
      private_D[0][0] = D[(32 * b0 + t0) * 1024 + (32 * b1 + t1)];
      private_D[0][1] = D[(32 * b0 + t0) * 1024 + (32 * b1 + t1 + 16)];
      for (int c2 = 0; c2 <= 1023; c2 += 32) {
        for (int c4 = t1; c4 <= 31; c4 += 16)
          shared_A[t0][c4] = A[(32 * b0 + t0) * 1024 + (c2 + c4)];
        for (int c4 = t1; c4 <= 31; c4 += 16)
          shared_B[t0][c4] = B[(32 * b1 + t0) * 1024 + (c2 + c4)];
        __syncthreads();
        for (int c3 = 0; c3 <= 31; c3 += 1) {
          private_D[0][0] += (((alpha) * shared_A[t0][c3]) * shared_B[t1][c3]);
          private_D[0][1] += (((alpha) * shared_A[t0][c3]) * shared_B[t1 + 16][c3]);
        }
        __syncthreads();
      }
      D[(32 * b0 + t0) * 1024 + (32 * b1 + t1)] = private_D[0][0];
      D[(32 * b0 + t0) * 1024 + (32 * b1 + t1 + 16)] = private_D[0][1];
    }
}
